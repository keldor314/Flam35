
#include <hip/hip_runtime.h>
﻿__shared__ unsigned int randStates[32];

__constant__ unsigned int cshift1[4] = { 6, 2, 13, 3 };
__constant__ unsigned int cshift2[4] = { 13, 27, 21, 12 };
__constant__ unsigned int cshift3[4] = { 18, 2, 7, 13 };
__constant__ unsigned int coffset[4] = { 4294967294, 4294967288, 4294967280, 4294967168 };

__shared__ unsigned int shift1[4];
__shared__ unsigned int shift2[4];
__shared__ unsigned int shift3[4];
__shared__ unsigned int offset[4];

__device__ unsigned int TausStep(unsigned int &z)
{
	int index = threadIdx.x & 3;
	unsigned int b = (((z << shift1[index]) ^ z) >> shift2[index]);
	return z = (((z & offset[index]) << shift3[index]) ^ b);
}

__device__ unsigned int randInt()
{
	TausStep(randStates[threadIdx.x & 31]);
	return (randStates[(threadIdx.x) & 31] ^ randStates[(threadIdx.x + 1) & 31] ^ randStates[(threadIdx.x + 2) & 31] ^ randStates[(threadIdx.x + 3) & 31]);
}

extern "C" __global__ void AtomicPointPoolTest(float4* pointPool, unsigned int* irandStates)
{
	if (threadIdx.x < 4)
	{
		shift1[threadIdx.x] = cshift1[threadIdx.x];
		shift2[threadIdx.x] = cshift2[threadIdx.x];
		shift3[threadIdx.x] = cshift3[threadIdx.x];
		offset[threadIdx.x] = coffset[threadIdx.x];
	}
	randStates[threadIdx.x] = irandStates[threadIdx.x + 32 * blockIdx.x];
	for (int n = 0; n < 256; n++)
	{
		unsigned int input = randInt() & 255;
		unsigned int output = randInt() & 255;
		pointPool[output] = pointPool[input];
	}
}
